/*************************************************************************
    > File Name: add.cu
    > Author: anryyang
    > Mail: anryyang@gmail.com 
    > Created Time: Mon 26 Feb 2018 04:53:27 PM SGT
 ************************************************************************/


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    int a, b, c;  // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    a = 2;
    b = 7;
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("%d\n", c);
    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}

