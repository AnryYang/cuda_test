
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE_BLOCK 256

__global__ void add(int n, int *a, int *b, int *c) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=start; i< n; i+=stride){
        c[i] = a[i] + b[i];
    }
}

void random_ints(int* x, int size)
{
    int i;
    for (i=0;i<size;i++) {
        x[i]=rand()%10;
    }
}

int main(void) 
{
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int n = 1<<20;
    int num_block = (n + SIZE_BLOCK - 1) / SIZE_BLOCK;
    int size = n * sizeof(int);
    
    // Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, n);
    b = (int *)malloc(size); random_ints(b, n);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    // Launch add() kernel on GPU with N blocks
    add<<<num_block,SIZE_BLOCK>>>(n, d_a, d_b, d_c);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // Cleanup

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
